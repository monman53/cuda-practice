#include <torch/torch.h>
#include <iostream>

void reduction2() {
    std::cout << "Hello world!" << std::endl;

    float *input;
    float *output;

    size_t size = 1000;
    size_t bytes = size * sizeof(float);

    hipMalloc(&input, bytes);
    hipMalloc(&output, bytes);

    auto dims = torch::IntArrayRef{1, int(size)};
    auto gpu_tensor = torch::from_blob(input, dims, torch::TensorOptions().device(torch::kCUDA));
    torch::Tensor tensor = torch::rand({2, 3});
    std::cout << tensor << std::endl;
}