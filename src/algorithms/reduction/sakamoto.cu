#include <common.hpp>
#include <algorithms/reduction.hpp>
#include <torch/torch.h>

class ReductionSakamoto : public Reduction {
public:
    std::string name() {
        return "Sakamoto";
    }

    void sum_1d(const float *x, size_t n, float *y) {
        y[0] = 0;
        for(size_t i=0;i<n;i++){
            y[0] += x[i];
        }
    };
};

void reduction() {
    std::cout << "Hello world!" << std::endl;

    ReductionSakamoto red;
    std::cout << red.name() << std::endl;

    // float *input;
    // float *output;

    // size_t size = 1000;
    // size_t bytes = size * sizeof(float);

    // hipMalloc(&input, bytes);
    // hipMalloc(&output, bytes);

    // auto dims = torch::IntArrayRef{1, int(size)};
    // auto gpu_tensor = torch::from_blob(input, dims, torch::TensorOptions().device(torch::kCUDA));
    torch::Tensor tensor = torch::rand({2, 3});
    std::cout << tensor << std::endl;
}